#include "hip/hip_runtime.h"

#include <cuspatial/error.hpp>
#include <cuspatial/shapefile_reader.hpp>
#include <cuspatial/spatial_join.hpp>

#include <cuspatial/point_quadtree.hpp>
#include <cuspatial/polygon_bounding_box.hpp>
#include <cuspatial/spatial_join.hpp>

#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/copying.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/reduction.hpp>
#include <cudf/types.hpp>
#include <cudf/null_mask.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <rmm/device_uvector.hpp>
#include "spatial_join_test_utility.cuh"
#include "spatial_join_test_utility.hpp"

std::unique_ptr<cudf::column> make_numeric_column(cudf::data_type type,
                                            cudf::size_type size,
                                            cudf::mask_state state,
                                            hipStream_t stream,
                                            rmm::mr::device_memory_resource* mr)
{
  //CUDF_FUNC_RANGE();
  //CUDF_EXPECTS(is_numeric(type), "Invalid, non-numeric type.");

  return std::make_unique<cudf::column>(type,
                                  size,
                                  rmm::device_buffer{size * cudf::size_of(type), stream, mr},
                                  create_null_mask(size, state, stream, mr),
                                  state_null_count(state, size),
                                  std::vector<std::unique_ptr<cudf::column>>{});
}

struct SpatialJoinNYCTaxiTest
{        
    uint32_t num_pnts=0;
    uint32_t num_quadrants=0;
    uint32_t num_pq_pairs=0;
    uint32_t num_pp_pairs=0;

    //point x/y on host
    double *h_pnt_x=nullptr,*h_pnt_y=nullptr;
    double *d_pnt_x=NULL,*d_pnt_y=NULL;
    uint32_t *h_point_indices=nullptr;

    //quadtree length/fpos
    uint32_t *h_qt_length=nullptr,*h_qt_fpos=nullptr;   

    //quadrant/polygon pairs
    uint32_t *h_pq_quad_idx=nullptr,*h_pq_poly_idx=nullptr;   
    
    //point/polygon pairs on device; shared between run_test and compute_mismatch
    //the life span of d_pp_pnt_idx/d_pp_poly_idx depends on pip_pair_tbl
    uint32_t *h_pp_pnt_idx=nullptr,*h_pp_poly_idx=nullptr;
    std::unique_ptr<cudf::column> col_pnt_x,col_pnt_y;
    std::unique_ptr<cudf::column> col_poly_fpos,col_poly_rpos,col_poly_x,col_poly_y;    
    
    hipStream_t stream=0;
    rmm::mr::device_memory_resource* mr=rmm::mr::get_current_device_resource();

    SBBox<double> setup_polygons(const char *file_name)
    {
        std::vector<std::unique_ptr<cudf::column>> polygon_columns = cuspatial::read_polygon_shapefile(file_name);
        std::cout<<"setup_polygons::polygon_columns="<<polygon_columns.size()<<std::endl;
        
        col_poly_fpos=std::move(polygon_columns.at(0));
        col_poly_rpos=std::move(polygon_columns.at(1));
        col_poly_x=std::move(polygon_columns.at(2));
        col_poly_y=std::move(polygon_columns.at(3));
        std::cout<<"setup_polygons::col_poly_fpos.size="<<col_poly_fpos->size()<<" col_poly_rpos.size="<<col_poly_rpos->size()<<std::endl;
        std::cout<<"setup_polygons::x.size="<<col_poly_x->size()<<" y.size="<<col_poly_y->size()<<std::endl;
        
        std::cout<<"col_poly_fpos"<<std::endl;
        thrust::device_ptr<const uint32_t> d_poly_fpos=thrust::device_pointer_cast(col_poly_fpos->view().begin<uint32_t>());
        thrust::copy(d_poly_fpos,d_poly_fpos+col_poly_fpos->size(),std::ostream_iterator<const uint32_t>(std::cout, " "));std::cout<<std::endl; 
  
        std::cout<<"col_poly_rpos"<<std::endl;
        thrust::device_ptr<const uint32_t> d_poly_rpos=thrust::device_pointer_cast(col_poly_rpos->view().begin<uint32_t>());
        thrust::copy(d_poly_rpos,d_poly_rpos+col_poly_rpos->size(),std::ostream_iterator<const uint32_t>(std::cout, " "));std::cout<<std::endl; 
               
        /*const double *x1_p=thrust::min_element(thrust::device,col_poly_x.begin<double>(),col_poly_x.end<double>());
        const double *x2_p=thrust::max_element(thrust::device,col_poly_x.begin<double>(),col_poly_x.end<double>());
        const double *y1_p=thrust::min_element(thrust::device,col_poly_y.begin<double>(),col_poly_y.end<double>());
        const double *y2_p=thrust::max_element(thrust::device,col_poly_y.begin<double>(),col_poly_y.end<double>());
        double x1,y1,x2,y2;
        hipMemcpy(&x1,x1_p, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&y1,y1_p, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&x2,x2_p, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&y2,y2_p, sizeof(double), hipMemcpyDeviceToHost);*/ 
        
        std::unique_ptr<cudf::scalar> x1_s=cudf::reduce(col_poly_x->view(), cudf::make_min_aggregation(),cudf::data_type{cudf::type_id::FLOAT64});
        std::unique_ptr<cudf::scalar> x2_s=cudf::reduce(col_poly_x->view(), cudf::make_max_aggregation(),cudf::data_type{cudf::type_id::FLOAT64});
        std::unique_ptr<cudf::scalar> y1_s=cudf::reduce(col_poly_y->view(), cudf::make_min_aggregation(),cudf::data_type{cudf::type_id::FLOAT64});
        std::unique_ptr<cudf::scalar> y2_s=cudf::reduce(col_poly_y->view(), cudf::make_max_aggregation(),cudf::data_type{cudf::type_id::FLOAT64});
        
        // auto const result_scalar = static_cast<cudf::scalar_type_t<decimalXX>*>(result.get());

        auto x1=static_cast<cudf::scalar_type_t<double>*>(x1_s.get())->value();
        auto y1=static_cast<cudf::scalar_type_t<double>*>(y1_s.get())->value();
        auto x2=static_cast<cudf::scalar_type_t<double>*>(x2_s.get())->value();
        auto y2=static_cast<cudf::scalar_type_t<double>*>(y2_s.get())->value();
  
        std::cout<<"x1="<<x1<<" x2="<<x2<<" y1="<<y1<<" y2="<<y2<<std::endl;
        
        return SBBox<double>(thrust::make_tuple(x1,y1), thrust::make_tuple(x2,y2));
    }

    SBBox<double> setup_points(const char * file_name, uint32_t first_n)
    {
        num_pnts=0;
        //read invidual data file  
        std::vector<uint32_t> len_vec;
        std::vector<double *> x_vec;
        std::vector<double *> y_vec;
        uint32_t num=0;
        
        FILE *fp=nullptr;
        if((fp=fopen(file_name,"r"))==nullptr)
        {
           std::cout<<"Failed to open point catalog file "<<file_name<<std::endl;
           exit(-2);          
        }
        while(!feof(fp))
        {
             char str[500];
             int n1=fscanf(fp,"%s",str);
             std::cout<<"processing point data file "<<str<<std::endl;
             double *tmp_x=nullptr,*tmp_y=nullptr;
             size_t temp_len=read_point_binary(str,tmp_x,tmp_y);
             assert(tmp_x!=nullptr && tmp_y!=nullptr);
             num++;
             len_vec.push_back(temp_len);
             x_vec.push_back(tmp_x);
             y_vec.push_back(tmp_y);
             if(first_n>0 && num>=first_n) break;
        }    
        fclose(fp);

        //prepare memory allocation
        for(uint32_t i=0;i<num;i++)
            num_pnts+=len_vec[i];
        uint32_t p=0;
        h_pnt_x=new double[num_pnts];
        h_pnt_y=new double[num_pnts];
        assert(h_pnt_x!=nullptr && h_pnt_y!=nullptr);
        
        //concatination
        for(uint32_t i=0;i<num;i++)
        {
            double *tmp_x=x_vec[i];
            double *tmp_y=y_vec[i];
            assert(tmp_x!=nullptr && tmp_y!=nullptr);
            int len=len_vec[i];
            std::copy(tmp_x,tmp_x+len,h_pnt_x+p);
            std::copy(tmp_y,tmp_y+len,h_pnt_y+p);
            p+=len;
            delete[] tmp_x;
            delete[] tmp_y;
        }
        assert(p==num_pnts);

        //compute the bbox of all points; outlier points may have irrational values
        //any points that do not fall within the Area of Interests (AOIs) will be assgin a special Morton code
        //AOI is user-defined and is passed to quadtree indexing and spatial join 
        double x1=*(std::min_element(h_pnt_x,h_pnt_x+num_pnts));
        double x2=*(std::max_element(h_pnt_x,h_pnt_x+num_pnts));
        double y1=*(std::min_element(h_pnt_y,h_pnt_y+num_pnts));
        double y2=*(std::max_element(h_pnt_y,h_pnt_y+num_pnts));
        std::cout<<"read_point_catalog: x_min="<<x1<<"  y_min="<<y1<<" x_max="<<x2<<" y_max="<<y2<<std::endl;

        //create x/y columns, expose their raw pointers to be used in run_test() and populate x/y arrays
        col_pnt_x = make_numeric_column( cudf::data_type{cudf::type_id::FLOAT64}, 
            num_pnts, cudf::mask_state::UNALLOCATED, stream, mr );      
        d_pnt_x=cudf::mutable_column_device_view::create(col_pnt_x->mutable_view(), stream)->data<double>();
        assert(d_pnt_x!=nullptr);
        HANDLE_CUDA_ERROR( hipMemcpy( d_pnt_x, h_pnt_x, num_pnts * sizeof(double), hipMemcpyHostToDevice ) );

        col_pnt_y = make_numeric_column( cudf::data_type{cudf::type_id::FLOAT64}, 
            num_pnts, cudf::mask_state::UNALLOCATED, stream, mr );      
        d_pnt_y=cudf::mutable_column_device_view::create(col_pnt_y->mutable_view(), stream)->data<double>();
        assert(d_pnt_y!=nullptr);    
        HANDLE_CUDA_ERROR( hipMemcpy( d_pnt_y, h_pnt_y, num_pnts * sizeof(double), hipMemcpyHostToDevice ) );
        
        return SBBox<double>(thrust::make_tuple(x1,y1), thrust::make_tuple(x2,y2));
    }

    void run_test(double x1,double y1,double x2,double y2,double scale,uint32_t num_level,uint32_t min_size)
    {
        timeval t0,t1,t2,t3,t4;

        gettimeofday(&t0, nullptr); 
        cudf::mutable_column_view pnt_x_view=col_pnt_x->mutable_view();
        cudf::mutable_column_view pnt_y_view=col_pnt_y->mutable_view();
        std::cout<<"run_test::num_pnts="<<col_pnt_x->size()<<std::endl;
        
        auto quadtree_pair =cuspatial::quadtree_on_points(pnt_x_view,pnt_y_view,x1,x2,y1,y2, scale,num_level, min_size,mr);       
        std::unique_ptr<cudf::table> quadtree_tbl=std::move(std::get<1>(quadtree_pair));
        std::unique_ptr<cudf::column> point_indices =std::move(std::get<0>(quadtree_pair));
        num_quadrants=quadtree_tbl->view().num_rows();
        std::cout<<"# of quadrants="<<num_quadrants<<std::endl;
        gettimeofday(&t1, nullptr);
        float quadtree_time=calc_time("quadtree_tbl constrution time=",t0,t1);

        //compute polygon bbox on GPU
         auto bbox_tbl=cuspatial::polygon_bounding_boxes(col_poly_fpos->view(),col_poly_rpos->view(),
            col_poly_x->view(),col_poly_y->view(),mr);
            
        gettimeofday(&t2, nullptr);
        float polybbox_time=calc_time("compute polygon bbox time=",t1,t2);
        std::cout<<"# of polygon bboxes="<<bbox_tbl->view().num_rows()<<std::endl;

        //spatial filtering
        const cudf::table_view quad_view=quadtree_tbl->view();
        const cudf::table_view bbox_view=bbox_tbl->view();
      
        std::unique_ptr<cudf::table>  pq_pair_tbl = 
        	cuspatial::join_quadtree_and_bounding_boxes(quad_view, bbox_view, x1, x2, y1, y2, scale, num_level,mr);

            
        gettimeofday(&t3, nullptr);
        float filtering_time=calc_time("spatial filtering time=",t2,t3);
        std::cout<<"# of polygon/quad pairs="<<pq_pair_tbl->view().num_rows()<<std::endl;

        //spatial refinement 
        std::unique_ptr<cudf::table> pip_pair_tbl = cuspatial::quadtree_point_in_polygon(pq_pair_tbl->view(),
                                                                     quadtree_tbl->view(),
                                                                     point_indices->view(),
                                                                     col_pnt_x->view(),
                                                                     col_pnt_y->view(),
                                                                     col_poly_fpos->view(),
                                                                     col_poly_rpos->view(),
                                                                     col_poly_x->view(),
                                                                     col_poly_y->view(),
                                                                     mr);
        gettimeofday(&t4, nullptr);
        float refinement_time=calc_time("spatial refinement time=",t3,t4);
        std::cout<<"# of polygon/point pairs="<<pip_pair_tbl->view().num_rows()<<std::endl;

        gettimeofday(&t1, nullptr);
        float gpu_time=calc_time("gpu end-to-end computing time=",t0,t1);

        //summierize runtimes
        float  runtimes[4]={quadtree_time,polybbox_time,filtering_time,refinement_time};
        const char  *msg_type[4]={"quadtree_time","polybbox_time","filtering_time","refinement_time"};
        float total_time=0;
        for(uint32_t i=0;i<4;i++)
        {
            std::cout<<msg_type[i]<<"= "<<runtimes[i]<<std::endl;
            total_time+=runtimes[i];
        }
        std::cout<<std::endl;
        std::cout<<"total_time="<<total_time<<std::endl;
        std::cout<<"gpu end-to-tend time"<<gpu_time<<std::endl;
        
        //copy back sorted points to CPU for verification
        //HANDLE_CUDA_ERROR( hipMemcpy(h_pnt_x, d_pnt_x,num_pnts * sizeof(double), hipMemcpyDeviceToHost ) );
        //HANDLE_CUDA_ERROR( hipMemcpy(h_pnt_y, d_pnt_y,num_pnts * sizeof(double), hipMemcpyDeviceToHost ) );

        const uint32_t * d_point_indices=point_indices->view().data<uint32_t>();
        h_point_indices=new uint32_t[num_pnts];
        HANDLE_CUDA_ERROR( hipMemcpy(h_point_indices, d_point_indices,num_pnts * sizeof(uint32_t), hipMemcpyDeviceToHost ) );
        
        //setup variables for verifications
        const uint32_t *d_qt_length=quadtree_tbl->view().column(3).data<uint32_t>();
        const uint32_t *d_qt_fpos=quadtree_tbl->view().column(4).data<uint32_t>();

        h_qt_length=new uint32_t[num_quadrants];
        h_qt_fpos=new uint32_t[num_quadrants];
        assert(h_qt_length!=nullptr && h_qt_fpos!=nullptr);

        HANDLE_CUDA_ERROR( hipMemcpy( h_qt_length, d_qt_length, num_quadrants * sizeof(uint32_t), hipMemcpyDeviceToHost) ); 
        HANDLE_CUDA_ERROR( hipMemcpy( h_qt_fpos, d_qt_fpos, num_quadrants * sizeof(uint32_t), hipMemcpyDeviceToHost) );

        num_pq_pairs=pq_pair_tbl->num_rows();
        const uint32_t * d_pq_poly_idx=pq_pair_tbl->view().column(0).data<uint32_t>();
        const uint32_t * d_pq_quad_idx=pq_pair_tbl->view().column(1).data<uint32_t>();

        h_pq_poly_idx=new uint32_t[num_pq_pairs];
        h_pq_quad_idx=new uint32_t[num_pq_pairs];
        assert(h_pq_poly_idx!=nullptr && h_pq_quad_idx!=nullptr);

        HANDLE_CUDA_ERROR( hipMemcpy( h_pq_poly_idx, d_pq_poly_idx, num_pq_pairs * sizeof(uint32_t), hipMemcpyDeviceToHost) ); 
        HANDLE_CUDA_ERROR( hipMemcpy( h_pq_quad_idx, d_pq_quad_idx, num_pq_pairs * sizeof(uint32_t), hipMemcpyDeviceToHost) );

        num_pp_pairs=pip_pair_tbl->num_rows();
        const uint32_t *d_pp_poly_idx=pip_pair_tbl->mutable_view().column(0).data<uint32_t>();
        const uint32_t *d_pp_pnt_idx=pip_pair_tbl->mutable_view().column(1).data<uint32_t>();

        h_pp_poly_idx=new uint32_t[num_pp_pairs];
        h_pp_pnt_idx=new uint32_t[num_pp_pairs];
        assert(h_pp_poly_idx!=nullptr && h_pp_pnt_idx!=nullptr);

        HANDLE_CUDA_ERROR( hipMemcpy( h_pp_poly_idx, d_pp_poly_idx, num_pp_pairs * sizeof(uint32_t), hipMemcpyDeviceToHost) ); 
        HANDLE_CUDA_ERROR( hipMemcpy( h_pp_pnt_idx, d_pp_pnt_idx, num_pp_pairs * sizeof(uint32_t), hipMemcpyDeviceToHost) );
    }
    
    void write_nyc_taxi(const char *file_name)
    {
        CUDF_EXPECTS(file_name!=NULL,"file_name can not be NULL");
        FILE *fp=fopen(file_name,"wb");
        CUDF_EXPECTS(fp!=NULL, "can not open file for output");
        
        CUDF_EXPECTS(fwrite(&(num_pnts),sizeof(uint32_t),1,fp)==1,"writting num_pnt failed");
        CUDF_EXPECTS(fwrite(&(num_quadrants),sizeof(uint32_t),1,fp)==1,"writting num_quadrants failed");
        CUDF_EXPECTS(fwrite(&(num_pq_pairs),sizeof(uint32_t),1,fp)==1,"writting num_pq_pairs failed");
        CUDF_EXPECTS(fwrite(&(num_pp_pairs),sizeof(uint32_t),1,fp)==1,"writting num_pp_pairs failed");
        
        CUDF_EXPECTS(fwrite(h_pnt_x,sizeof(double),num_pnts,fp)==num_pnts,"writting h_pnt_x failed");
        CUDF_EXPECTS(fwrite(h_pnt_y,sizeof(double),num_pnts,fp)==num_pnts,"writting h_pnt_y failed");
        CUDF_EXPECTS(fwrite(h_point_indices,sizeof(uint32_t),num_pnts,fp)==num_pnts,"writting h_point_indices failed");
        
        CUDF_EXPECTS(fwrite(h_qt_length,sizeof(uint32_t),num_quadrants,fp)==num_quadrants,"writting h_qt_length failed");
        CUDF_EXPECTS(fwrite(h_qt_fpos,sizeof(uint32_t),num_quadrants,fp)==num_quadrants,"writting h_qt_fpos failed");
        
        CUDF_EXPECTS(fwrite(h_pq_quad_idx,sizeof(uint32_t),num_pq_pairs,fp)==num_pq_pairs,"writting h_pq_quad_idx failed");
        CUDF_EXPECTS(fwrite(h_pq_poly_idx,sizeof(uint32_t),num_pq_pairs,fp)==num_pq_pairs,"writting h_pq_poly_idx failed");
        
        CUDF_EXPECTS(fwrite(h_pp_poly_idx,sizeof(uint32_t),num_pp_pairs,fp)==num_pp_pairs,"writting h_pp_poly_idx failed");
        CUDF_EXPECTS(fwrite(h_pp_pnt_idx,sizeof(uint32_t),num_pp_pairs,fp)==num_pp_pairs,"writting h_pp_pnt_idx failed");
    }    
};


int main()
{
    SpatialJoinNYCTaxiTest test;
    
    const char* env_p = std::getenv("CUSPATIAL_DATA");
    CUDF_EXPECTS(env_p!=nullptr,"CUSPATIAL_DATA environmental variable must be set");
    
    const uint32_t num_level=15;
    const uint32_t min_size=512;
    const uint32_t first_n=12; 

    std::cout<<"loading NYC taxi pickup locations..........."<<std::endl;
    
    //from https://www1.nyc.gov/site/tlc/about/tlc-trip-record-data.page; 
    //pickup/drop-off locations are extracted and the lon/lat coordiates are converted to epsg:2263 projection
        
    //a catalog file is simply a collection of invidual binary data files with a pre-defined structure
    //each line repersents a data file, e.g., pickup+drop-off locations for a month
    std::string catalog_filename=std::string(env_p)+std::string("2009.cat"); 
    std::cout<<"Using catalog file "<<catalog_filename<<std::endl;
    test.setup_points(catalog_filename.c_str(),first_n);

    std::cout<<"loading NYC polygon data..........."<<std::endl;

    enum POLYID {taxizone_id=0,cd_id,ct_id};    
    POLYID sel_id=taxizone_id;

    const char * shape_files[]={"taxi_zones.shp","nycd_11a_av/nycd.shp","nyct2000_11a_av/nyct2000.shp"};
    
    const char * bin_files[]={"nyc_taxizone_2009_1.bin","nyc_cd_2009_12.bin","nyc_ct_2009_12.bin"};
 
    std::cout<<"loading NYC polygon data..........."<<std::endl;

    std::string shape_filename=std::string(env_p)+std::string(shape_files[sel_id]); 
    
    std::cout<<"Using shapefile "<<shape_filename<<std::endl;

    SBBox<double> aoi=test.setup_polygons(shape_filename.c_str());

    double poly_x1=thrust::get<0>(aoi.first);
    double poly_y1=thrust::get<1>(aoi.first);
    double poly_x2=thrust::get<0>(aoi.second);
    double poly_y2=thrust::get<1>(aoi.second);
    
    double width=poly_x2-poly_x1;
    double height=poly_y2-poly_y1;
    double length=(width>height)?width:height;
    double scale=length/((1<<num_level)+2);
    double bbox_x1=poly_x1-scale;
    double bbox_y1=poly_y1-scale;
    double bbox_x2=poly_x2+scale; 
    double bbox_y2=poly_y2+scale;
    printf("Area of Interests: length=%15.10f scale=%15.10f\n",length,scale);

    std::cout<<"running test on NYC taxi trip data..........."<<std::endl;
    test.run_test(bbox_x1,bbox_y1,bbox_x2,bbox_y2,scale,num_level,min_size);
    
    write_nyc_taxi(bin_files[sel_id]);
    
    return(0); 
}

