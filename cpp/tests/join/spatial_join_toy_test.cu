#include "hip/hip_runtime.h"
#include <random>

#include <cuspatial/error.hpp>
#include <cuspatial/shapefile_reader.hpp>
#include <cuspatial/spatial_join.hpp>

#include <cuspatial/point_quadtree.hpp>
#include <cuspatial/polygon_bounding_box.hpp>
#include <cuspatial/spatial_join.hpp>

#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/copying.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/reduction.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <rmm/device_uvector.hpp>

#include "spatial_join_test_utility.cuh"
#include "spatial_join_test_utility.hpp"


std::unique_ptr<column> make_numeric_column(data_type type,
                                            size_type size,
                                            mask_state state,
                                            hipStream_t stream,
                                            rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  CUDF_EXPECTS(is_numeric(type), "Invalid, non-numeric type.");

  return std::make_unique<column>(type,
                                  size,
                                  rmm::device_buffer{size * cudf::size_of(type), stream, mr},
                                  create_null_mask(size, state, stream, mr),
                                  state_null_count(state, size),
                                  std::vector<std::unique_ptr<column>>{});
}

template <typename T>
inline auto generate_points(std::vector<std::vector<T>> const &quads, uint32_t points_per_quad)
{
  std::vector<T> point_x(quads.size() * points_per_quad);
  std::vector<T> point_y(quads.size() * points_per_quad);

  std::seed_seq seed{time(0)};
  std::mt19937 g(seed);
  
  for (uint32_t i = 0, pos = 0; i < quads.size(); i++, pos += points_per_quad) {
    std::uniform_real_distribution<> dist_x (quads[i][0], quads[i][1]);
    std::uniform_real_distribution<> dist_y (quads[i][0], quads[i][1]);

    std::generate(point_x.begin() + pos, point_x.begin() + pos + points_per_quad, [&]() mutable {
      return dist_x(g);
    });

    std::generate(point_y.begin() + pos, point_y.begin() + pos + points_per_quad, [&]() mutable {
      return dist_y(g);
    });
  }
  return std::make_pair(std::move(point_x), std::move(point_y));
}


struct SpatialJoinNYCTaxiTest
{        
    uint32_t num_pnts=0;
    uint32_t num_quadrants=0;
    uint32_t num_pq_pairs=0;
    uint32_t num_pp_pairs=0;

    //point x/y on host
    double *h_pnt_x=nullptr,*h_pnt_y=nullptr;
    uint32_t *h_point_indices=nullptr;

    //quadtree length/fpos
    uint32_t *h_qt_length=nullptr,*h_qt_fpos=nullptr;   

    //quadrant/polygon pairs
    uint32_t *h_pq_quad_idx=nullptr,*h_pq_poly_idx=nullptr;   
    
    //point/polygon pairs on device; shared between run_test and compute_mismatch
    //the life span of d_pp_pnt_idx/d_pp_poly_idx depends on pip_pair_tbl
    uint32_t *h_pp_pnt_idx=nullptr,*h_pp_poly_idx=nullptr;
    std::unique_ptr<cudf::column> col_pnt_x,col_pnt_y;
    std::unique_ptr<cudf::column> col_poly_fpos,col_poly_rpos,col_poly_x,col_poly_y;    
    
    hipStream_t stream=0;
    rmm::mr::device_memory_resource* mr=rmm::mr::get_current_device_resource();

    SBBox<double> setup_polygons(const char *file_name)
    {
        std::vector<std::unique_ptr<cudf::column>> polygon_columns = cuspatial::read_polygon_shapefile(file_name);
        std::cout<<"setup_polygons::polygon_columns="<<polygon_columns.size()<<std::endl;
        
        col_poly_fpos=std::move(polygon_columns.at(0));
        col_poly_rpos=std::move(polygon_columns.at(1));
        col_poly_x=std::move(polygon_columns.at(2));
        col_poly_y=std::move(polygon_columns.at(3));
        std::cout<<"setup_polygons::col_poly_fpos.size="<<col_poly_fpos->size()<<" col_poly_rpos.size="<<col_poly_rpos->size()<<std::endl;
        std::cout<<"setup_polygons::x.size="<<col_poly_x->size()<<" y.size="<<col_poly_y->size()<<std::endl;
        
        std::cout<<"col_poly_fpos"<<std::endl;
        thrust::device_ptr<const uint32_t> d_poly_fpos=thrust::device_pointer_cast(col_poly_fpos->view().begin<uint32_t>());
        thrust::copy(d_poly_fpos,d_poly_fpos+col_poly_fpos->size(),std::ostream_iterator<const uint32_t>(std::cout, " "));std::cout<<std::endl; 
  
        std::cout<<"col_poly_rpos"<<std::endl;
        thrust::device_ptr<const uint32_t> d_poly_rpos=thrust::device_pointer_cast(col_poly_rpos->view().begin<uint32_t>());
        thrust::copy(d_poly_rpos,d_poly_rpos+col_poly_rpos->size(),std::ostream_iterator<const uint32_t>(std::cout, " "));std::cout<<std::endl; 

        std::cout<<"col_poly_x"<<std::endl;
        thrust::device_ptr<const double> d_poly_xx=thrust::device_pointer_cast(col_poly_x->view().begin<double>());
        thrust::copy(d_poly_xx,d_poly_xx+col_poly_x->size(),std::ostream_iterator<const double>(std::cout, " "));std::cout<<std::endl;

        std::cout<<"col_poly_y"<<std::endl;
        thrust::device_ptr<const double> d_poly_yy=thrust::device_pointer_cast(col_poly_y->view().begin<double>());
        thrust::copy(d_poly_yy,d_poly_yy+col_poly_y->size(),std::ostream_iterator<const double>(std::cout, " "));std::cout<<std::endl;
 
        /*const double *x1_p=thrust::min_element(thrust::device,col_poly_x.begin<double>(),col_poly_x.end<double>());
        const double *x2_p=thrust::max_element(thrust::device,col_poly_x.begin<double>(),col_poly_x.end<double>());
        const double *y1_p=thrust::min_element(thrust::device,col_poly_y.begin<double>(),col_poly_y.end<double>());
        const double *y2_p=thrust::max_element(thrust::device,col_poly_y.begin<double>(),col_poly_y.end<double>());
        double x1,y1,x2,y2;
        hipMemcpy(&x1,x1_p, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&y1,y1_p, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&x2,x2_p, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&y2,y2_p, sizeof(double), hipMemcpyDeviceToHost);*/ 
        
        std::unique_ptr<cudf::scalar> x1_s=cudf::reduce(col_poly_x->view(), cudf::make_min_aggregation(),cudf::data_type{cudf::type_id::FLOAT64});
        std::unique_ptr<cudf::scalar> x2_s=cudf::reduce(col_poly_x->view(), cudf::make_max_aggregation(),cudf::data_type{cudf::type_id::FLOAT64});
        std::unique_ptr<cudf::scalar> y1_s=cudf::reduce(col_poly_y->view(), cudf::make_min_aggregation(),cudf::data_type{cudf::type_id::FLOAT64});
        std::unique_ptr<cudf::scalar> y2_s=cudf::reduce(col_poly_y->view(), cudf::make_max_aggregation(),cudf::data_type{cudf::type_id::FLOAT64});
        
        // auto const result_scalar = static_cast<cudf::scalar_type_t<decimalXX>*>(result.get());

        auto x1=static_cast<cudf::scalar_type_t<double>*>(x1_s.get())->value();
        auto y1=static_cast<cudf::scalar_type_t<double>*>(y1_s.get())->value();
        auto x2=static_cast<cudf::scalar_type_t<double>*>(x2_s.get())->value();
        auto y2=static_cast<cudf::scalar_type_t<double>*>(y2_s.get())->value();
  
        std::cout<<"x1="<<x1<<" x2="<<x2<<" y1="<<y1<<" y2="<<y2<<std::endl;
        
        return SBBox<double>(thrust::make_tuple(x1,y1), thrust::make_tuple(x2,y2));
    }

    SBBox<double> setup_points(uint32_t const min_size)
    {
 
        std::vector<std::vector<double>> quads{{0, 2, 0, 2},
                                       {3, 4, 0, 1},
                                       {2, 3, 1, 2},
                                       {4, 6, 0, 2},
                                       {3, 4, 2, 3},
                                       {2, 3, 3, 4},
                                       {6, 7, 2, 3},
                                       {7, 8, 3, 4},
                                       {0, 4, 4, 8}};
   
        auto host_points = generate_points<double>(quads, min_size);
        this->num_pnts=std::get<0>(host_points).size();
        this->h_pnt_x        =new double[this->num_pnts];
        this->h_pnt_y        =new double[this->num_pnts];
    
        auto h_x_vec=std::get<0>(host_points);
        auto h_y_vec=std::get<1>(host_points);
        std::copy(h_x_vec.begin(),h_x_vec.end(),this->h_pnt_x );
        std::copy(h_y_vec.begin(),h_y_vec.end(),this->h_pnt_y );

        //compute the bbox of all points; outlier points may have irrational values
        //any points that do not fall within the Area of Interests (AOIs) will be assgin a special Morton code
        //AOI is user-defined and is passed to quadtree indexing and spatial join 
        double x1=*(std::min_element(h_pnt_x,h_pnt_x+this->num_pnts));
        double x2=*(std::max_element(h_pnt_x,h_pnt_x+this->num_pnts));
        double y1=*(std::min_element(h_pnt_y,h_pnt_y+this->num_pnts));
        double y2=*(std::max_element(h_pnt_y,h_pnt_y+this->num_pnts));
        std::cout<<"read_point: x_min="<<x1<<"  y_min="<<y1<<" x_max="<<x2<<" y_max="<<y2<<std::endl;

        //create x/y columns, expose their raw pointers to be used in run_test() and populate x/y arrays
        this->col_pnt_x = make_numeric_column( cudf::data_type{cudf::type_id::FLOAT64}, 
            this->num_pnts, cudf::mask_state::UNALLOCATED, stream, mr );      
        double *d_pnt_x=cudf::mutable_column_device_view::create(col_pnt_x->mutable_view(), stream)->data<double>();
        assert(d_pnt_x!=nullptr);
        HANDLE_CUDA_ERROR( hipMemcpy( d_pnt_x, h_pnt_x, this->num_pnts * sizeof(double), hipMemcpyHostToDevice ) );

        this->col_pnt_y = make_numeric_column( cudf::data_type{cudf::type_id::FLOAT64}, 
            this->num_pnts, cudf::mask_state::UNALLOCATED, stream, mr );      
        double *d_pnt_y=cudf::mutable_column_device_view::create(col_pnt_y->mutable_view(), stream)->data<double>();
        assert(d_pnt_y!=nullptr);    
        HANDLE_CUDA_ERROR( hipMemcpy( d_pnt_y, h_pnt_y, this->num_pnts * sizeof(double), hipMemcpyHostToDevice ) );
        
        return SBBox<double>(thrust::make_tuple(x1,y1), thrust::make_tuple(x2,y2));
    }

    void run_test(double x1,double y1,double x2,double y2,double scale,uint32_t num_level,uint32_t min_size)
    {
        timeval t0,t1,t2,t3,t4;

        gettimeofday(&t0, nullptr); 
        cudf::mutable_column_view pnt_x_view=col_pnt_x->mutable_view();
        cudf::mutable_column_view pnt_y_view=col_pnt_y->mutable_view();
        std::cout<<"run_test::num_pnts="<<col_pnt_x->size()<<std::endl;
        
        auto quadtree_pair =cuspatial::quadtree_on_points(pnt_x_view,pnt_y_view,x1,x2,y1,y2, scale,num_level, min_size,this->mr);       
        std::unique_ptr<cudf::table> quadtree_tbl=std::move(std::get<1>(quadtree_pair));
        std::unique_ptr<cudf::column> point_indices =std::move(std::get<0>(quadtree_pair));
        this->num_quadrants=quadtree_tbl->view().num_rows();
        std::cout<<"# of quadrants="<<this->num_quadrants<<std::endl;
        gettimeofday(&t1, nullptr);
        float quadtree_time=calc_time("quadtree_tbl constrution time=",t0,t1);

        //compute polygon bbox on GPU
         auto bbox_tbl=cuspatial::polygon_bounding_boxes(col_poly_fpos->view(),col_poly_rpos->view(),
            col_poly_x->view(),col_poly_y->view(),this->mr);
            
        gettimeofday(&t2, nullptr);
        float polybbox_time=calc_time("compute polygon bbox time=",t1,t2);
        std::cout<<"# of polygon bboxes="<<bbox_tbl->view().num_rows()<<std::endl;

        //spatial filtering
        const cudf::table_view quad_view=quadtree_tbl->view();
        const cudf::table_view bbox_view=bbox_tbl->view();
      
        std::unique_ptr<cudf::table>  pq_pair_tbl = 
        	cuspatial::join_quadtree_and_bounding_boxes(quad_view, bbox_view, x1, x2, y1, y2, scale, num_level,this->mr);

        uint32_t num_pq_pairs=pq_pair_tbl->num_rows();
 
        thrust::host_vector<uint32_t> pq_poly_id(num_pq_pairs);
        thrust::host_vector<uint32_t> pq_quad_id(num_pq_pairs);
        
        HANDLE_CUDA_ERROR( hipMemcpy(pq_poly_id.data(), pq_pair_tbl->get_column(0).view().data<uint32_t>(),num_pq_pairs* sizeof(uint32_t), hipMemcpyDeviceToHost ) );
        HANDLE_CUDA_ERROR( hipMemcpy(pq_quad_id.data(), pq_pair_tbl->get_column(1).view().data<uint32_t>(),num_pq_pairs* sizeof(uint32_t), hipMemcpyDeviceToHost ) );
               
       printf("num_pq_pairs=%d\n",num_pq_pairs);
        for(uint32_t i=0;i<num_pq_pairs;i++)
        {
		   printf("%d, %d, %d\n",i,pq_poly_id[i],pq_quad_id[i]);
        }
            
        gettimeofday(&t3, nullptr);
        float filtering_time=calc_time("spatial filtering time=",t2,t3);
        std::cout<<"# of polygon/quad pairs="<<pq_pair_tbl->view().num_rows()<<std::endl;

        //spatial refinement 
        std::unique_ptr<cudf::table> pip_pair_tbl = cuspatial::quadtree_point_in_polygon(pq_pair_tbl->view(),
                                                                     quadtree_tbl->view(),
                                                                     point_indices->view(),
                                                                     col_pnt_x->view(),
                                                                     col_pnt_y->view(),
                                                                     col_poly_fpos->view(),
                                                                     col_poly_rpos->view(),
                                                                     col_poly_x->view(),
                                                                     col_poly_y->view(),
                                                                     this->mr);

        gettimeofday(&t4, nullptr);
        float refinement_time=calc_time("spatial refinement time=",t3,t4);
        std::cout<<"# of polygon/point pairs="<<pip_pair_tbl->view().num_rows()<<std::endl;

        gettimeofday(&t1, nullptr);
        float gpu_time=calc_time("gpu end-to-end computing time=",t0,t1);

        //summierize runtimes
        float  runtimes[4]={quadtree_time,polybbox_time,filtering_time,refinement_time};
        const char  *msg_type[4]={"quadtree_time","polybbox_time","filtering_time","refinement_time"};
        float total_time=0;
        for(uint32_t i=0;i<4;i++)
        {
            std::cout<<msg_type[i]<<"= "<<runtimes[i]<<std::endl;
            total_time+=runtimes[i];
        }
        std::cout<<std::endl;
        std::cout<<"total_time="<<total_time<<std::endl;
        std::cout<<"gpu end-to-tend time"<<gpu_time<<std::endl;
        
        const uint32_t * d_point_indices=point_indices->view().data<uint32_t>();
        this->h_point_indices=new uint32_t[this->num_pnts];
        HANDLE_CUDA_ERROR( hipMemcpy(h_point_indices, d_point_indices,this->num_pnts * sizeof(uint32_t), hipMemcpyDeviceToHost ) );
                
        //setup variables for verifications
        const uint32_t *d_qt_length=quadtree_tbl->view().column(3).data<uint32_t>();
        const uint32_t *d_qt_fpos=quadtree_tbl->view().column(4).data<uint32_t>();

        this->h_qt_length=new uint32_t[this->num_quadrants];
        this->h_qt_fpos=new uint32_t[this->num_quadrants];
        assert(this->h_qt_length!=nullptr && this->h_qt_fpos!=nullptr);

        HANDLE_CUDA_ERROR( hipMemcpy( h_qt_length, d_qt_length, this->num_quadrants * sizeof(uint32_t), hipMemcpyDeviceToHost) ); 
        HANDLE_CUDA_ERROR( hipMemcpy( h_qt_fpos, d_qt_fpos, this->num_quadrants * sizeof(uint32_t), hipMemcpyDeviceToHost) );

        this->num_pq_pairs=pq_pair_tbl->num_rows();
        const uint32_t * d_pq_poly_idx=pq_pair_tbl->view().column(0).data<uint32_t>();
        const uint32_t * d_pq_quad_idx=pq_pair_tbl->view().column(1).data<uint32_t>();

        this->h_pq_poly_idx=new uint32_t[num_pq_pairs];
        this->h_pq_quad_idx=new uint32_t[num_pq_pairs];
        assert(this->h_pq_poly_idx!=nullptr && this->h_pq_quad_idx!=nullptr);

        HANDLE_CUDA_ERROR( hipMemcpy( this->h_pq_poly_idx, d_pq_poly_idx, num_pq_pairs * sizeof(uint32_t), hipMemcpyDeviceToHost) ); 
        HANDLE_CUDA_ERROR( hipMemcpy( this->h_pq_quad_idx, d_pq_quad_idx, num_pq_pairs * sizeof(uint32_t), hipMemcpyDeviceToHost) );

        this->num_pp_pairs=pip_pair_tbl->num_rows();
        const uint32_t *d_pp_poly_idx=pip_pair_tbl->mutable_view().column(0).data<uint32_t>();
        const uint32_t *d_pp_pnt_idx=pip_pair_tbl->mutable_view().column(1).data<uint32_t>();

        this->h_pp_poly_idx=new uint32_t[num_pp_pairs];
        this->h_pp_pnt_idx=new uint32_t[num_pp_pairs];
        assert(this->h_pp_poly_idx!=nullptr && this->h_pp_pnt_idx!=nullptr);

        HANDLE_CUDA_ERROR( hipMemcpy( this->h_pp_poly_idx, d_pp_poly_idx, num_pp_pairs * sizeof(uint32_t), hipMemcpyDeviceToHost) ); 
        HANDLE_CUDA_ERROR( hipMemcpy( this->h_pp_pnt_idx, d_pp_pnt_idx, num_pp_pairs * sizeof(uint32_t), hipMemcpyDeviceToHost) );
    }
    
    void write_points_bin(const char *file_name)
    {
        CUDF_EXPECTS(file_name!=NULL,"file_name can not be NULL");
        FILE *fp=fopen(file_name,"wb");
        CUDF_EXPECTS(fp!=NULL, "can not open file for output");
        
        uint32_t tb=0;
        CUDF_EXPECTS(fwrite(&(this->num_pnts),sizeof(uint32_t),1,fp)==1,"writting num_pnt failed");
        CUDF_EXPECTS(fwrite(&(this->num_quadrants),sizeof(uint32_t),1,fp)==1,"writting num_quadrants failed");
        CUDF_EXPECTS(fwrite(&(this->num_pq_pairs),sizeof(uint32_t),1,fp)==1,"writting num_pq_pairs failed");
        CUDF_EXPECTS(fwrite(&(this->num_pp_pairs),sizeof(uint32_t),1,fp)==1,"writting num_pp_pairs failed");
        tb+=4*sizeof(uint32_t);
        
        CUDF_EXPECTS(fwrite(this->h_pnt_x,sizeof(double),this->num_pnts,fp)==this->num_pnts,"writting h_pnt_x failed");
        CUDF_EXPECTS(fwrite(this->h_pnt_y,sizeof(double),this->num_pnts,fp)==this->num_pnts,"writting h_pnt_y failed");
        CUDF_EXPECTS(fwrite(this->h_point_indices,sizeof(uint32_t),this->num_pnts,fp)==this->num_pnts,"writting h_point_indices failed");
        tb+=(this->num_pnts*(sizeof(double)*2+sizeof(uint32_t)));
        thrust::copy(this->h_pnt_x,this->h_pnt_x+this->num_pnts,std::ostream_iterator<double>(std::cout, " "));std::cout<<std::endl;    
        thrust::copy(this->h_pnt_y,this->h_pnt_y+this->num_pnts,std::ostream_iterator<double>(std::cout, " "));std::cout<<std::endl;   
        thrust::copy(this->h_point_indices,this->h_point_indices+this->num_pnts,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;   
        
        CUDF_EXPECTS(fwrite(this->h_qt_length,sizeof(uint32_t),this->num_quadrants,fp)==this->num_quadrants,"writting h_qt_length failed");
        CUDF_EXPECTS(fwrite(this->h_qt_fpos,sizeof(uint32_t),this->num_quadrants,fp)==this->num_quadrants,"writting h_qt_fpos failed");
        tb+=(this->num_quadrants*sizeof(uint32_t)*2);
        thrust::copy(this->h_qt_length,this->h_qt_length+this->num_quadrants,std::ostream_iterator<double>(std::cout, " "));std::cout<<std::endl;    
        thrust::copy(this->h_qt_fpos,this->h_qt_fpos+this->num_quadrants,std::ostream_iterator<double>(std::cout, " "));std::cout<<std::endl;   
        
        CUDF_EXPECTS(fwrite(this->h_pq_quad_idx,sizeof(uint32_t),this->num_pq_pairs,fp)==this->num_pq_pairs,"writting h_pq_quad_idx failed");
        CUDF_EXPECTS(fwrite(this->h_pq_poly_idx,sizeof(uint32_t),this->num_pq_pairs,fp)==this->num_pq_pairs,"writting h_pq_poly_idx failed");
        tb+=(this->num_pq_pairs*sizeof(uint32_t)*2);
        thrust::copy(this->h_pq_quad_idx,this->h_pq_quad_idx+this->num_pq_pairs,std::ostream_iterator<double>(std::cout, " "));std::cout<<std::endl;    
        thrust::copy(this->h_pq_poly_idx,this->h_pq_poly_idx+this->num_pq_pairs,std::ostream_iterator<double>(std::cout, " "));std::cout<<std::endl;   
        
        
        CUDF_EXPECTS(fwrite(this->h_pp_poly_idx,sizeof(uint32_t),this->num_pp_pairs,fp)==this->num_pp_pairs,"writting h_pp_poly_idx failed");
        CUDF_EXPECTS(fwrite(this->h_pp_pnt_idx,sizeof(uint32_t),this->num_pp_pairs,fp)==this->num_pp_pairs,"writting h_pp_pnt_idx failed");
        tb+=(this->num_pp_pairs*sizeof(uint32_t)*2);
        thrust::copy(this->h_pp_poly_idx,this->h_pp_poly_idx+this->num_pp_pairs,std::ostream_iterator<double>(std::cout, " "));std::cout<<std::endl;    
        thrust::copy(this->h_pp_pnt_idx,this->h_pp_pnt_idx+this->num_pp_pairs,std::ostream_iterator<double>(std::cout, " "));std::cout<<std::endl;   

        for(uint32_t i=0;i<this->num_pp_pairs;i++)
        {
           uint32_t pid=this->h_point_indices[this->h_pp_pnt_idx[i]];
           printf("%d,%10.5f, %10.5f, %d\n",i,this->h_pnt_x[pid],this->h_pnt_y[pid],this->h_pp_poly_idx[i]);
	}
  
        printf("total bytes=%d\n",tb);  
    }    
};


int main()
{
    SpatialJoinNYCTaxiTest test;
    
    const char* env_p = std::getenv("CUSPATIAL_DATA");
    CUDF_EXPECTS(env_p!=nullptr,"CUSPATIAL_DATA environmental variable must be set");
    
    std::cout<<"loading point data..........."<<std::endl;
    uint32_t const max_depth{3};
    uint32_t const min_size{400};
    double const scale{1.0};
    test.setup_points(min_size);        

    std::cout<<"loading polygon data..........."<<std::endl;
    std::string shape_filename=std::string(env_p)+std::string("quad_test_ply.shp");     
    std::cout<<"Using shapefile "<<shape_filename<<std::endl;
    SBBox<double> aoi=test.setup_polygons(shape_filename.c_str());

    //verify all polygon vertices (x,y) in the shapefile are between [0.0,8.0) and [0.0,8.0) 
    
    double poly_x1=thrust::get<0>(aoi.first);
    double poly_y1=thrust::get<1>(aoi.first);
    double poly_x2=thrust::get<0>(aoi.second);
    double poly_y2=thrust::get<1>(aoi.second);
    
    printf("x1=%10.5f y1=%10.5f x2=%10.5f y2=%10.5f\n",poly_x1,poly_y1,poly_x2,poly_y2);
    
    std::cout<<"running test on toy data..........."<<std::endl;
    test.run_test(0.0,0.0,8.0,8.0,scale,max_depth,min_size);
    
    test.write_points_bin("toy_points.bin");
    
    return(0);
}

